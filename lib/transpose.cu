#include "hip/hip_runtime.h"
#include "kernel.h"

// native

__global__ void transpose_native(float* input, float* output, int M, int N) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < M && col < N) {
        output[col * M + row] = input[row * N + col];
    }
}

// memory coalescing
__global__ void transpose_mc(float* input, float* output, int M, int N) {
    const int BM = 16;
    const int BN = 16;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ float tile[BM][BN];

    // g2s
    int row = by * BM + ty;
    int col = bx * BN + tx;
    if (row < M && col < N) {
        tile[ty][tx] = input[row * N + col];
    }

    __syncthreads();

    // write back to global memory, do transpose
    col = bx * BN + tx;
    row = by * BM + ty;
    if (row < M && col < N) {
        output[row * M + col] = tile[tx][ty];
    }
}

// memory coalescing wtih bank free
__global__ void transpose_mc_bank_free(float* input, float* output, int M, int N) {
    const int BM = 16;
    const int BN = 16;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ float tile[BM][BN+1];

    // g2s
    int row = by * BM + ty;
    int col = bx * BN + tx;
    if (row < M && col < N) {
        tile[ty][tx] = input[row * N + col];
    }

    __syncthreads();

    // write back to global memory, do transpose
    col = bx * BN + tx;
    row = by * BM + ty;
    if (row < M && col < N) {
        output[row * M + col] = tile[tx][ty];
    }
}



template<const int M, const int N, void (*F)(float*, float*, int, int)>
void call_kernel() {
    float* input = (float*)malloc(M * N * sizeof(float));
    float* output = (float*)malloc(M * N * sizeof(float));

    for (int i = 0; i < M * N; i++) {
        input[i] = i;
    }

    float* d_input;
    float* d_output;

    hipMalloc(&d_input, M * N * sizeof(float));
    hipMalloc(&d_output, M * N * sizeof(float));

    hipMemcpy(d_input, input, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    F<<<grid, block>>>(d_input, d_output, M, N);
    hipEventRecord(stop);

    hipMemcpy(output, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    // caculate bandwidth
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float bandwidth = (M * N * sizeof(float) * 2) / milliseconds / 1e6;
    std::cout << "Bandwidth: " << bandwidth << " GB/s" << std::endl;


    // for (int i = 0; i < N; i++) {
    //     for (int j = 0; j < M; j++) {
    //         std::cout << output[i * M + j] << " ";
    //         if (j == M - 1) std::cout << std::endl;
    //     }
    // }

    free(input);
    free(output);

    hipFree(d_input);
    hipFree(d_output);
}






int main() {

    call_kernel<20000, 15000, transpose_native>();
    call_kernel<20000, 15000, transpose_mc>();
    call_kernel<20000, 15000, transpose_mc_bank_free>();

    return 0;
}